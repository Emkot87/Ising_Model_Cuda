
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

struct timeval startwtime, endwtime;
double seq_time;

__global__ void Ising(int* G, int* L,int n, int threads,int bSize){
  int index = threadIdx.x + blockIdx.x * threads;
  int s = ceil(n/(float) bSize);
  int k = index/s;
  int f = index%s;
      
  for(int i = (k * bSize) ; i < (k+1) * bSize && i < n; i++){
    for(int j = (f * bSize) ; j < (f+1) * bSize && j < n; j++ ){
      int x = G[(i-1+n)%n*n+j] + G[i*n+(j-1+n)%n] + G[i*n+j] + G[(i+1)%n*n+j] + G[i*n+(j+1)%n];
      L[i*n+j] = (x>0) - (x<0);
    }
  }
    
      
}

int main(int argc, char* argv[]){

    int n ,k;
    FILE *fptr;
    srand((unsigned int)time(NULL));

  
    if(argc < 5){
      printf("We need 4 arguments, number of iterations, size, block size and threads per Gpu block");
      return 0;
	  }

    k = (int) strtol(argv[1],NULL,10);
    n = (int) strtol(argv[2],NULL,10);

    int* F = (int*)malloc(n * n * sizeof(int));
    int* L = (int*)malloc(n * n * sizeof(int));

    int *d_F, *d_L;
    hipMalloc(&d_F, n * n * sizeof(int));
    hipMalloc(&d_L, n * n * sizeof(int));

    // reads file
    fptr = fopen("input.bin","rb");
    fread(F, sizeof(int), n * n, fptr);
    fclose(fptr);

    hipMemcpy(d_F, F, n * n * sizeof(int), hipMemcpyHostToDevice);

    

    int bSize = (int) strtol(argv[3],NULL,10);
    int threadsPerBlock = (int) strtol(argv[4],NULL,10);
    int s = ceil(n/(float) bSize);
    int blocks = ceil((s * s)/(float)threadsPerBlock);
    printf("blocks %d\n", blocks);
    int threads =  ceil((s * s)/(float)blocks);
    printf("threads %d\n", threads);
    
   gettimeofday (&startwtime, NULL);
    
    for(int i = 0; i < k ; i++ ){
        Ising<<<blocks,threads>>>(d_F,d_L,n,threads,bSize);
        int* temp = d_F;
        d_F = d_L;
        d_L = temp;
        hipDeviceSynchronize();
    }

    hipDeviceSynchronize();
    gettimeofday (&endwtime, NULL);
		seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
    printf("\n\n-=-=-=-=-=-=-=-+++total time %f+++-=-=-=-=-=-=-=-=-=-\n\n",seq_time);

    hipMemcpy(F, d_F, n * n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(L, d_L, n * n * sizeof(int), hipMemcpyDeviceToHost);


    int* checker = (int*)malloc(n * n * sizeof(int));
    fptr = fopen("output.bin","rb");
    fread(checker, sizeof(int), n * n, fptr);
    fclose(fptr);

    int allOk = 0;

    for(int i = 0 ; i < n ; i++){
      for(int k = 0 ; k < n ; k++){
        if(checker[i*n + k] != F[i*n + k]){
          allOk++;
        }
      }
    }

    if(allOk){
      printf("Something went wrong %d times\n",allOk);
    }
    else{
      printf("Everything is correct\n");
    }


    // Free everything
    free(L);
    free(F);
    hipFree(d_F);
    hipFree(d_L);

}
