
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

struct timeval startwtime, endwtime;
double seq_time;

__global__ void Ising(int* G, int* L,int n, int threads){
      int index = threadIdx.x + blockIdx.x * threads;
      int i = index/n;
      int j = index%n;
      int x = G[(i-1+n)%n*n+j] + G[i*n+(j-1+n)%n] + G[i*n+j] + G[(i+1)%n*n+j] + G[i*n+(j+1)%n];
      L[i*n+j] = (x>0) - (x<0);
}

int main(int argc, char* argv[]){

    int n ,k;
    FILE *fptr;
    srand((unsigned int)time(NULL));

    // read arguments
    if(argc < 4){
		printf("We need 3 arguments, number of iterations, size and threads per Gpu block");
		return 0;
	}

    k = (int) strtol(argv[1],NULL,10);
    n = (int) strtol(argv[2],NULL,10);
    int threadsPerBlock = (int) strtol(argv[3],NULL,10);

    int* F = (int*)malloc(n * n * sizeof(int));
    int* L = (int*)malloc(n * n * sizeof(int));

    int *d_F, *d_L;
    hipMalloc(&d_F, n * n * sizeof(int));
    hipMalloc(&d_L, n * n * sizeof(int));

    // read the input file V0 made
    fptr = fopen("input.bin","rb");
    fread(F, sizeof(int), n * n, fptr);
    fclose(fptr);

    hipMemcpy(d_F, F, n * n * sizeof(int), hipMemcpyHostToDevice);

    
   
    int blocks = ceil((n * n)/(float)threadsPerBlock);
    printf("blocks %d\n", blocks);
    int threads =  ceil((n * n)/(float)blocks);
    printf("threads %d\n", threads);

    gettimeofday (&startwtime, NULL);

    for(int i = 0; i < k ; i++ ){
        Ising<<<blocks,threads>>>(d_F,d_L,n,threads);
        int* temp = d_F;
        d_F = d_L;
        d_L = temp;
        hipDeviceSynchronize();
    }
    
    hipDeviceSynchronize();
    gettimeofday (&endwtime, NULL);
		seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6 + endwtime.tv_sec - startwtime.tv_sec);
		printf("\n\n-=-=-=-=-=-=-=-+++total time %f+++-=-=-=-=-=-=-=-=-=-\n\n",seq_time);

    hipMemcpy(F, d_F, n * n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(L, d_L, n * n * sizeof(int), hipMemcpyDeviceToHost);

 
    int* checker = (int*)malloc(n * n * sizeof(int));
    fptr = fopen("output.bin","rb");
    fread(checker, sizeof(int), n * n, fptr);
    fclose(fptr);

    int allOk = 0;

    for(int i = 0 ; i < n ; i++){
      for(int k = 0 ; k < n ; k++){
        if(checker[i*n + k] != F[i*n + k]){
          allOk++;
        }
      }
    }

    if(allOk){
      printf("Something went wrong %d times\n",allOk);
    }
    else{
      printf("Everything is correct\n");
    }


    // Free everything
    free(L);
    free(F);
    hipFree(d_F);
    hipFree(d_L);

}